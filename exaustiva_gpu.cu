#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/copy.h>

using namespace std;

#define MAXCATEGORIES 10

struct Movie {
    int id;
    int start;
    int end;
    int category;
};



void read_movies_booking(int size, vector<Movie> &book) {
    /**
     * Recebe um inteiro com a quantidade de filmes a ser lida
     * Recebe um vetor de book a ser preenchido com os filmes
     */
    for (int i = 0; i < size; i++) {
        Movie m;
        int start_time, end_time;
        m.id = i;
        cin >> start_time;
        cin >> end_time;

        if (end_time < start_time) {
            end_time = 23; // acaba na última hora do dia
        }

        int category = 0;
        cin >> category;

        // HARD FIX -> Alinha categoria com o índice dos arrays
        m.category = category - 1;

        m.start = start_time;
        m.end = end_time;

        book.push_back(m);
    }
}

long int get_exhibition_time(Movie movie) {
    /**
     * Recebe um vetor de inteiros target a ser preenchido com os horários de exibição
     * Recebe um vetor de filmes movies
     */
    int exhibition_time = 0;
    for (int j = movie.start; j <= movie.end; j++) {
        exhibition_time |= (1 << j);
    }

    return exhibition_time;
}

struct count_watched_movies {

    int n;
    int k;

    long int *exhibition_time;
    int *categories;

    int *categories_max_size;

    count_watched_movies(
        int n_,
        int k_,
        long int *exhibition_time_,
        int *categories_,
        int *categories_max_size_) :

        n(n_),
        k(k_),
        exhibition_time(exhibition_time_),
        categories(categories_),
        categories_max_size(categories_max_size_) {}

    __device__
    int operator()(const int &allocation) const
    {
        int watched_movies_count = 0;

        int categories_watched[MAXCATEGORIES];
        for (int i = 0; i < MAXCATEGORIES; i++) {
            categories_watched[i] = 0;
        }

        long int available_exhibition_time = 0;

        // para cada um dos filmes indicados como 1 (a assistir), checa se é possível assistir
        for (int i = 0; i < n; i++) {
            // caso o filme tenha sido alocado (1), analisa validade
            if (allocation & (1 << i)) {
                // checa se há disponibilidade de categoria
                if (categories_watched[categories[i]] < categories_max_size[categories[i]]) {
                    // checa se o horário de exibição está disponível
                    long int has_override = available_exhibition_time & exhibition_time[i];
                    if (has_override == 0)  {
                        watched_movies_count++;
                        categories_watched[categories[i]]++;
                        available_exhibition_time |= exhibition_time[i];
                    } else {

                        return -1;
                    }
                } else {
                    return -2;
                }
            }
        }

        return watched_movies_count;
    }

};
int main(int argc, char* argv[]){

    int n = 0;
    int k = 0;

    cin >> n;
    cin >> k;

    // Armazena o máximo de filmes para cada categoria
    thrust::host_vector<int> h_categories_max_size(k, 0);
    for (int i = 0; i < k; i++) {
        int max_size = 0;
        cin >> max_size;
        h_categories_max_size[i] = max_size;
    }

    // Lê os filmes e os armazena em um vetor
    vector<Movie> movies;
    read_movies_booking(n, movies);

    // Vamos criar alguns vetores: Categoria e Horários de Exibição

    thrust::host_vector<int> h_categories(n);
    thrust::host_vector<long int> h_exhibition_time(n);

    // preenche o vetor h_exhibition com um inteiro que representa o binário de horas em que o filme é exibido
    for (int i=0; i<n; i++){
        h_categories[i] = movies[i].category;
        h_exhibition_time[i] = get_exhibition_time(movies[i]);
    }


    // Vamos obter um vetor de inteiros, cuja transformação em bits representa cada filme a ser assistido
    
    thrust::device_vector<int> d_allocations(pow(2, n));
    thrust::sequence(d_allocations.begin(), d_allocations.end());

    // Vamos transformar os vetores para GPU
    thrust::device_vector<int> d_categories = h_categories;
    thrust::device_vector<long int> d_exhibition_time = h_exhibition_time;
    thrust::device_vector<int> d_categories_max_size = h_categories_max_size;

    // Vamos aplicar a função de verificação para testar cada possibilidade, se é válida e contar quantos filmes foram assistidos.
    thrust::transform(
        d_allocations.begin(),
        d_allocations.end(),
        d_allocations.begin(),
        count_watched_movies(
            n,
            k,
            thrust::raw_pointer_cast(d_exhibition_time.data()),
            thrust::raw_pointer_cast(d_categories.data()),
            thrust::raw_pointer_cast(d_categories_max_size.data())
        )
    );

    int max_watched_movies = thrust::reduce(d_allocations.begin(), d_allocations.end(), 0, thrust::maximum<int>());

    cout << "Máximo de filmes assistidos: " << max_watched_movies << endl;


    return 0;
}
