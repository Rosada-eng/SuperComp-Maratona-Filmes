#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

using namespace std;

#define MAXCATEGORIES 10

struct Movie {
    int id;
    int start;
    int end;
    int category;
};

void read_max_categories_amount(int k, vector<int> &cat_max_size) {

    for (int i = 0; i < k; i++) {
        int max_size = 0;
        cin >> max_size;
        cat_max_size.push_back(max_size);
    }
}

void read_movies_booking(int size, vector<Movie> &book) {
    /**
     * Recebe um inteiro com a quantidade de filmes a ser lida
     * Recebe um vetor de book a ser preenchido com os filmes
     */
    for (int i = 0; i < size; i++) {
        Movie m;
        int start_time, end_time;
        m.id = i;
        cin >> start_time;
        cin >> end_time;

        if (end_time < start_time) {
            end_time = 23; // acaba na última hora do dia
        }

        int category = 0;
        cin >> category;

        // HARD FIX -> Alinha categoria com o índice dos arrays
        m.category = category - 1;

        m.start = start_time;
        m.end = end_time;

        book.push_back(m);
    }
}

int get_exhibition_time(Movie movie) {
    /**
     * Recebe um vetor de inteiros target a ser preenchido com os horários de exibição
     * Recebe um vetor de filmes movies
     */
    int exhibition_time = 0;
    for (int j = movie.start; j <= movie.end; j++) {
        exhibition_time |= (1 << j);
    }

    return exhibition_time;
}

struct count_watched_movies {

    int n;
    int k;

    int *exhibition_time;
    int *categories;

    int *categories_max_size;

    count_watched_movies(
        int n_,
        int k_,
        int *exhibition_time_,
        int *categories_,
        int *categories_max_size_) :

        n(n_),
        k(k_),
        exhibition_time(exhibition_time_),
        categories(categories_),
        categories_max_size(categories_max_size_) {}

    __device__
    int operator()(const int &allocation) 
    {
        int watched_movies_count = 0;
        int categories_watched[MAXCATEGORIES];
        for (int i = 0; i < MAXCATEGORIES; i++) {
            categories_watched[i] = 0;
        }
        int available_exhibition_time = 0;

        // para cada um dos filmes indicados como 1 (a assistir), checa se é possível assistir
        for (int i = 0; i < n; i++) {
            if (allocation & (1 << i)) {
                // checa se há disponibilidade de categoria
                if (categories_watched[categories[i]] +1 <= categories_max_size[categories[i]]) {
                    // checa se o horário de exibição está disponível
                    if (!(available_exhibition_time & exhibition_time[i] != 0))  {
                        watched_movies_count++;
                        categories_watched[categories[i]]++;
                        available_exhibition_time |= exhibition_time[i];
                    } else {

                        return 0;
                    }
                } else {
                    return 0;
                }
            }
        }

        return watched_movies_count;
    }

};
int main(int argc, char* argv[]){

    int n = 0;
    int k = 0;

    cin >> n;
    cin >> k;

    // Armazena o máximo de filmes para cada categoria
    vector<int> categories_max_size(k, 0);
    read_max_categories_amount(k, categories_max_size);

    // Lê os filmes e os armazena em um vetor
    vector<Movie> movies(n);

    // Vamos criar alguns vetores: Categoria e Horários de Exibição

    thrust::host_vector<int> h_categories(n);
    thrust::host_vector<int> h_exhibition_time(n);

    // preenche o vetor h_exhibition com um inteiro que representa o binário de horas em que o filme é exibido
    for (int i=0; i<n; i++){
        h_exhibition_time[i] = get_exhibition_time(movies[i]);
    }

    // Vamos obter um vetor de inteiros, cuja transformação em bits representa cada filme a ser assistido
    
    thrust::device_vector<int> d_allocations(pow(2, n));
    thrust::sequence(d_allocations.begin(), d_allocations.end());

    // Vamos transformar os vetores para GPU
    thrust::device_vector<int> d_categories = h_categories;
    thrust::device_vector<int> d_exhibition_time = h_exhibition_time;

    // Vamos aplicar a função de verificação para testar cada possibilidade, se é válida e contar quantos filmes foram assistidos.
    thrust::transform(
        d_allocations.begin(),
        d_allocations.end(),
        d_allocations.begin(),
        count_watched_movies(
            n,
            k,
            thrust::raw_pointer_cast(d_exhibition_time.data()),
            thrust::raw_pointer_cast(d_categories.data()),
            thrust::raw_pointer_cast(categories_max_size.data())
        )
    );


    int max_watched_movies = thrust::reduce(d_allocations.begin(), d_allocations.end(), 0, thrust::maximum<int>());

    cout << "Máximo de filmes assistidos: " << max_watched_movies << endl;


    return 0;
}
